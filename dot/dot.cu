#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <random>
#include <hipblas.h>
#include <hiprand.h>
#include "dot.cuh"

void cpu_dot(const float *a ,const float *b,float *c,int n  ){
    for(int i = 0; i < n; i++){
        *c += a[i] * b[i];
    }
}


// Wrapper function
void dot_wrapper(const float* a, const float* b, int n) {

    float *c1,*c2;
    hipMalloc(&c1, sizeof(float));
    hipMalloc(&c2, sizeof(float));
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer
    hipEventRecord(start);
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    // Launch kernel
    gpu_dot<<<numBlocks, blockSize>>>(a,b, c1, n);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;


    // Start the timer


    // Perform gemv operation using CUBLAS
    float alpha = 1.0;

    float beta = 0.0;
    hipblasHandle_t handle; // Declare the CUBLAS handle

    // Initialize the CUBLAS library
    hipblasCreate(&handle);

    hipEventRecord(start);
    // Perform the matrix-vector multiplication using CUBLAS
    hipblasSdot(handle, n, a, 1, b, 1, c2);

    // Destroy the CUBLAS handle
    hipblasDestroy(handle);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUBLAS execution time: " << milliseconds << " ms" << std::endl;
    
    float *h_c3 = new float;
    float *h_a = new float[n];
    float *h_b = new float[n];
    hipMemcpy(h_a, a, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_b, b, n * sizeof(float), hipMemcpyDeviceToHost);
    cpu_dot(h_a,h_b,h_c3,n);
    
    float *h_c1 = new float;
    float *h_c2 = new float;
    hipMemcpy(h_c1, c1, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_c2, c2, sizeof(float), hipMemcpyDeviceToHost);
    // Check the correctness of the result
    if (abs(*h_c1 - *h_c2) > 1e-4) {
        std::cout<< h_c1 << " " << h_c2 <<" "<<h_c3<<std::endl;
        std::cout << "Result is incorrect!" << std::endl;
    }else {
        std::cout << "Result is correct!" << std::endl;
    }
    // Free memory
    delete[] h_c1;
    delete[] h_c2;
    hipFree(c1);
    hipFree(c2);
}

// Data generation function
void generateData(float *a,float *b, int n) {
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerateUniform(gen, a, n);
    hiprandGenerateUniform(gen, b, n);

    hiprandDestroyGenerator(gen);
}

int main() {
    int n = 128; // Number of columns
    float *a, *b;
    hipMalloc(&a, n * sizeof(float));
    hipMalloc(&b, n * sizeof(float));

    // Generate random data
    generateData(a,b, n);
    // hipMemcpy(A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(x, h_x, n * sizeof(float), hipMemcpyHostToDevice);

    // Perform gemv operation
    dot_wrapper(a, b, n);

    // Print the result

    return 0;
}