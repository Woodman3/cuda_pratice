#include "hip/hip_runtime.h"
#include"check.cuh"
#include<iostream>
#include<stdio.h>
#define N 10

using namespace std;
template<typename T>
__global__ void fun(T* A,T* B,T* C){
    int i = threadIdx.x;
    C[i]=A[i]+B[i];
    printf("wtf%d\n",i);

}

int main(){
    int *h_A,*h_B,*h_C;
    h_A=(int*)malloc(N*sizeof(int));
    h_B=(int*)malloc(N*sizeof(int));
    h_C=(int*)malloc(N*sizeof(int));
    int *d_A,*d_B,*d_C;
    for(int i=0;i<N;i++){
        h_A[i]=i;
        h_B[i]=i+4;
        h_C[i]=3;
    } 
    CHECK(hipMalloc((void**)&d_A,N*sizeof(int)));
    CHECK(hipMalloc((void**)&d_B,N*sizeof(int)));
    CHECK(hipMalloc((void**)&d_C,N*sizeof(int)));
    CHECK(hipMemcpy(d_A,h_A,N*sizeof(int),::hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B,h_B,N*sizeof(int),::hipMemcpyHostToDevice));
    fun<int><<<1,N>>>(d_A,d_B,d_C);
    CHECK(hipMemcpy(h_C,d_C,N*sizeof(int),::hipMemcpyDeviceToHost));
    CHECK(hipGetLastError());
    hipDeviceSynchronize();
    cout<<h_C[0]<<endl;
    hipFree(d_A);hipFree(d_B);hipFree(d_C); 
}