
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <random>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#define OFFSET(row,col,R) (((row)*(R))+(col))
// Gemv function
// x: n x 1 vector
// A: m x n matrix
// y: m x 1 vector
__global__ void gemv(const float* A, const float* x, float* y, int m, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < m) {
        float temp = 0.0;
        for (int i = 0; i < n; i++) {
            temp += A[OFFSET(idx,i,n)] * x[i];
        }
        y[idx]=temp;
    }
}

void cup_gemv(const float* A, const float* x, float* y, int m, int n){
    for(int i=0;i<m;i++){
        y[i]=0.0;
        for(int j=0;j<n;j++){
            y[i]+=A[OFFSET(i,j,n)]*x[j];
        }
    }
}


// Wrapper function
void gemvWrapper(const float* A, const float* x, int m, int n) {

    float *y1,*y2;
    hipMalloc(&y1, m * sizeof(float));
    hipMalloc(&y2, m * sizeof(float));
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer
    hipEventRecord(start);
    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;
    // Launch kernel
    gemv<<<numBlocks, blockSize>>>(A, x, y1, m, n);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;


    // Start the timer


    // Perform gemv operation using CUBLAS
    float alpha = 1.0;

    float beta = 0.0;
    hipblasHandle_t handle; // Declare the CUBLAS handle

    // Initialize the CUBLAS library
    hipblasCreate(&handle);

    hipEventRecord(start);
    // Perform the matrix-vector multiplication using CUBLAS
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha, A, m, x, 1, &beta, y2, 1);

    // Destroy the CUBLAS handle
    hipblasDestroy(handle);

    // Stop the timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "CUBLAS execution time: " << milliseconds << " ms" << std::endl;
    
    float *h_y3 = new float[m];
    float *h_A = new float[m*n];
    float *h_x = new float[n];
    hipMemcpy(h_A, A, m * n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_x, x, n * sizeof(float), hipMemcpyDeviceToHost);
    cup_gemv(h_A,h_x,h_y3,m,n);
    
    float *h_y1 = new float[m];
    float *h_y2 = new float[m];
    hipMemcpy(h_y1, y1, m * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y2, y2, m * sizeof(float), hipMemcpyDeviceToHost);
    // Check the correctness of the result
    bool correct = true;
    for (int i = 0; i < m; i++) {
        if (abs(h_y1[i] - h_y3[i]) > 1e-5) {
            std::cout<< h_y1[i] << " " << h_y2[i] <<" "<<h_y3[i]<<" "<<i<< std::endl;
            
            std::cout << "Result is incorrect!" << std::endl;
            correct = false;
            break;
        }
    }
    if (correct)
        std::cout << "Result is correct!" << std::endl;
    // Free memory
    delete[] h_y1;
    delete[] h_y2;
    hipFree(y1);
    hipFree(y2);
}

// Data generation function
void generateData(float* A, float* x, int m, int n) {
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

    hiprandGenerateUniform(gen, A, m * n);
    hiprandGenerateUniform(gen, x, n);

    hiprandDestroyGenerator(gen);
}

int main() {
    int m = 512; // Number of rows
    int n = 32; // Number of columns

    // Allocate memory for matrices and vectors
    float *A,*x ;
    hipMalloc(&A, m * n * sizeof(float));
    hipMalloc(&x, n * sizeof(float));

    // Generate random data
    generateData(A, x, m, n);

    // Perform gemv operation
    gemvWrapper(A, x, m, n);

    // Print the result


    return 0;
}