
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>

template <unsigned int warpSize=32>
__forceinline__ __device__ float warp_reduce(float val) {
    for(int offset = warpSize>>1 ; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

template<unsigned int threadNum = 256>
__global__ void reduce_kernel(float* input, float* output, int size) {
    __shared__ float shared[threadNum];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        shared[tid] = input[i];
    }
    else {
        shared[tid] = 0;
    }

    __syncthreads();

    for (unsigned int stride = blockDim.x>>1; stride >= 32; stride >>= 1) {
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
        __syncthreads();
    }

    float val = shared[tid];
    val = warp_reduce(val);

    if (tid == 0) {
        atomicAdd(output, val);
    }
}

float reduce(float* input, int size) {
    float* deviceInput;
    float* deviceOutput;

    hipMalloc((void**)&deviceInput, size * sizeof(float));
    hipMalloc((void**)&deviceOutput, sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(deviceInput, input, size * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipEventRecord(start);

    reduce_kernel<<<blocksPerGrid, threadsPerBlock>>>(deviceInput, deviceOutput, size);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time: %f ms\n", milliseconds);

    float output;
    hipMemcpy(&output, deviceOutput, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(deviceInput);
    hipFree(deviceOutput);

    return output;
}

float generateTestData(float* input, int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(0.0, 1.0);
    float r=0.0;
    for (int i = 0; i < size; i++) {
        input[i] = dis(gen);
        r += input[i]; 
    }
    return r;
}
int main() {
    int size = 1<<25;
    float* input = new float[size];
    float ans = generateTestData(input, size);

    float result = reduce(input, size);

    printf("Reduced value: %f, right is %f\n", result, ans);

    return 0;
}