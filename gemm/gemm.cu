
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#define OFFSET(row,col,R) (((row)*(R))+(col))

using namespace std;

typedef double tt;

template<typename T>
__global__ void gemm(T *a,T *b,T *c,int n,int k,int m){
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int bx=blockIdx.x;
    int by=blockIdx.y;
    int bdx=blockDim.x;
    int bdy=blockDim.y;
    int x=bdx*bx+tx;
    int y=bdy*by+ty;
    if(x>=n||y>=m)
        return;
    int id=OFFSET(x,y,m);
    T count=0;
    for(int i=0;i<k;i++){
        count+=a[OFFSET(x,i,k)]*b[OFFSET(i,y,m)];
    }
    c[id]=count;

}

int main(){
    int m,k,n;
    ifstream fin;
    fin.open("./test");
    fin>>m>>k>>n;
    tt *h_a = new tt[n*k];
    tt *h_b = new tt[k*m];
    tt *h_c = new tt[n*m];
    tt *right = new tt[n*m];
    tt *d_a,*d_b,*d_c;
    for(int i=0;i<n*k;i++)
        fin>>h_a[i];
    for(int i=0;i<k*m;i++)
        fin>>h_b[i];
    for(int i=0;i<n*m;i++)
        fin>>right[i]; 
    hipMalloc(&d_a,m*k*sizeof(tt));
    hipMalloc(&d_b,k*n*sizeof(tt));
    hipMalloc(&d_c,m*n*sizeof(tt));
    hipMemcpy(d_a,h_a,m*k*sizeof(tt),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,k*n*sizeof(tt),hipMemcpyHostToDevice);
    int blockx=16;
    int blocky=16;
    dim3 BlockRange(blockx,blocky);
    dim3 GridRange(n/blockx+1,m/blocky+1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gemm<tt><<<GridRange,BlockRange>>>(d_a,d_b,d_c,m,k,n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << " ms" << endl;
    hipMemcpy(h_c,d_c,m*n*sizeof(tt),hipMemcpyDeviceToHost); 

    for(int i=0;i<m*n;i++){
        if(abs(right[i]-h_c[i])>0.01){
            cout<<"wrong"<<right[i]<<" "<<h_c[i]<<endl;
        }
    }
    cout<<"all is right!"<<endl;
    return 0;
}