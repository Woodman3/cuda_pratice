
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#define OFFSET(row,col,R) ((row*R)+col)

using namespace std;

typedef double tt;

template<typename T>
__global__ void gemm(T *a,T *b,T *c,int m,int k,int n){
    int tx=threadIdx.x;
    int ty=threadIdx.y;
    int bx=blockIdx.x;
    int by=blockIdx.y;
    int bdx=blockDim.x;
    int bdy=blockDim.y;
    int x=bdx*bx+tx;
    int y=bdy*by+ty;
    if(x>=m||y>=n)
        return;
    int id=OFFSET(x,y,n);
    T count=0;
    for(int i=0;i<k;i++){
        count+=a[OFFSET(x,i,k)]*b[OFFSET(i,y,n)];
    }
    c[id]=count;

}

int main(){
    int m,k,n;
    ifstream fin;
    fin.open("./test");
    fin>>m>>k>>n;
    tt *h_a = new tt[m*k];
    tt *h_b = new tt[k*n];
    tt *h_c = new tt[m*n];
    tt *right = new tt[m*n];
    tt *d_a,*d_b,*d_c;
    for(int i=0;i<m*k;i++)
        fin>>h_a[i];
    for(int i=0;i<k*n;i++)
        fin>>h_b[i];
    for(int i=0;i<m*n;i++)
        fin>>right[i]; 
    hipMalloc(&d_a,m*k*sizeof(tt));
    hipMalloc(&d_b,k*n*sizeof(tt));
    hipMalloc(&d_c,m*n*sizeof(tt));
    hipMemcpy(d_a,h_a,m*k*sizeof(tt),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,k*n*sizeof(tt),hipMemcpyHostToDevice);
    int blockx=16;
    int blocky=16;
    dim3 BlockRange(blockx,blocky);
    dim3 GridRange(m/blockx+1,n/blocky+1);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gemm<tt><<<GridRange,BlockRange>>>(d_a,d_b,d_c,m,k,n);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time: " << milliseconds << " ms" << endl;
    hipMemcpy(h_c,d_c,m*n*sizeof(tt),hipMemcpyDeviceToHost); 

    for(int i=0;i<m*n;i++){
        if(abs(right[i]-h_c[i])>0.01){
            cout<<"wrong"<<right[i]<<" "<<h_c[i]<<endl;
        }
    }
    cout<<"all is right!"<<endl;
    return 0;
}